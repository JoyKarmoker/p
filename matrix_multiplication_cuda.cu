
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;

__global__ void matrix_multiplication(int *A, int *B, int *C, int *DP){
    int N = DP[0];
    int M = DP[1]; 
    int P = DP[2];
    int number_of_matrix = DP[3]; 
    int number_of_core = DP[4];

    
    int i = threadIdx.x;
    int start_matrix_index = (number_of_matrix/ number_of_core) * i;
    int end_matrix_index = (number_of_matrix / number_of_core) * (i + 1);
    if(i == number_of_core - 1) end_matrix_index = number_of_matrix;

    for(int x = start_matrix_index; x < end_matrix_index; x++){
        int add_A = x * N * M;
        int add_B = x * M * P;
        int add_C = x * N * P;
        for(int i = 0; i < N; i++){
            for(int j = 0; j < P; j++){
                int sum = 0;
                for(int k = 0; k < M; k++){
                    sum += A[ add_A + i * M + k] * B[add_B + k * P + j];
                }
                C[add_C + i * P + j] = sum;
            }
        }
    }
}
int main(int argc , char *argv[]){
    cout << "Enter the dimenssion of the matrix, N, M , P: " << endl;
    int N , M , P;
    cin >> N >> M >> P;
    cout << "Enter the number of Matrix: " << endl;
    int number_of_matrix;
    cin >> number_of_matrix;
    int *A, *B, *C;
    int a_element = number_of_matrix * N * M;
    int b_element = number_of_matrix * M * P;
    int c_element = number_of_matrix * N * P;

    A = new int[a_element];
    B = new int[b_element];
    C = new int[c_element];

    for(int i = 0; i < a_element; i++){
        A[i] = 1;
    }
    for(int i = 0; i < b_element; i++){
        B[i] = 1;
    }
    int *DA, *DB, *DC;
    hipMalloc(&DA, a_element * sizeof(int));
    hipMalloc(&DB, b_element * sizeof(int));
    hipMalloc(&DC, c_element * sizeof(int));
    
    hipMemcpy(DA, A, a_element * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(DB, B, b_element * sizeof(int) , hipMemcpyHostToDevice);
  

    cout << "Enter the number of core you want to use." << endl;
    int number_of_core = 2;
    cin >> number_of_core;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);

    int parameter[] ={N , M , P, number_of_matrix, number_of_core};
    int *DP;
    hipMalloc(&DP, 5 * sizeof(int));
    hipMemcpy(DP, parameter, 5 * sizeof(int), hipMemcpyHostToDevice);

    matrix_multiplication <<< 1 , number_of_core >>> (DA, DB, DC, DP);

    hipDeviceSynchronize();

    hipEventRecord(end);
    hipEventSynchronize(end);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, end);
    cout << "Time taken to the gpu: with " << number_of_core << " is : " << milliseconds << " miliseconds." << endl;

    hipMemcpy(C, DC, c_element * sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++){
        for(int j = 0; j < P; j++){
            cout << C[i * P + j] << " ";
        }
        cout << endl;
    }
    
    return 0;

}